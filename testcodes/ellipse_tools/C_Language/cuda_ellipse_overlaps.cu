#include "hip/hip_runtime.h"
#include "cuda_ellipse_overlaps.cuh"

#include <vector>

#define PI 3.14159265358979
using std::vector;

__device__ inline void CalculateRangeAtY(double *elpparm, double y, double *x1, double *x2)
{
    double A, B, C, D, E, F;
    A = elpparm[0], B = elpparm[1], C = elpparm[2];
    D = elpparm[3], E = elpparm[4], F = elpparm[5];

    double Delta = pow(B*y + D, 2) - A*(C*y*y + 2 * E*y + F);

    if (Delta < 0)
        *x1 = -10, *x2 = -20;
    else
    {
        double t1, t2;
        t1 = (-(B*y + D) - sqrt(Delta)) / A;
        t2 = (-(B*y + D) + sqrt(Delta)) / A;

        if (t2 < t1)
        {
            double tmp = t1;
            t1 = t2;
            t2 = tmp;
        }

        *x1 = t1;
        *x2 = t2;
    }

}


__device__ inline void ELPShape2Equation(double *elpshape, double *outparms)
{
    double xc, yc, a, b, theta;
    xc = elpshape[0], yc = elpshape[1], a = elpshape[2]/2, b = elpshape[3]/2, theta = elpshape[4];

    double parm[6];

    parm[0] = cos(theta)*cos(theta) / (a*a) + pow(sin(theta), 2) / (b*b);
    parm[1] = -(sin(2 * theta)*(a*a - b*b)) / (2 * a*a*b*b);
    parm[2] = pow(cos(theta), 2) / (b*b) + pow(sin(theta), 2) / (a*a);
    parm[3] = (-a*a*xc*pow(sin(theta), 2) + a*a*yc*sin(2 * theta) / 2) / (a*a*b*b) - (xc*pow(cos(theta), 2) + yc*sin(2 * theta) / 2) / (a*a);
    parm[4] = (-a*a*yc*pow(cos(theta), 2) + a*a*xc*sin(2 * theta) / 2) / (a*a*b*b) - (yc*pow(sin(theta), 2) + xc*sin(2 * theta) / 2) / (a*a);
    parm[5] = pow(xc*cos(theta) + yc*sin(theta), 2) / (a*a) + pow(yc*cos(theta) - xc*sin(theta), 2) / (b*b) - 1;

    double k = parm[0] * parm[2] - parm[1] * parm[1];

    for (int i = 0; i < 6; i++)
        outparms[i] = parm[i] / sqrt(fabs(k));

}





__device__ inline void CalculateOverlap(double *elp1, double *elp2, double *_ration)
{

    /*
    for(int i=0; i< 5;i++)
    {
        std::cout<<elp1[i]<<std::endl;

    }
    for(int i=0; i< 5;i++)
    {
        std::cout<<elp2[i]<<std::endl;

    }
    */

    double parm1[6], parm2[6];
    ELPShape2Equation(elp1, parm1);
    ELPShape2Equation(elp2, parm2);

    double y1_min, y1_max, y2_min, y2_max, y_min, y_max;
    y1_min = elp1[1] - fmax(elp1[2], elp1[3]); y1_max = elp1[1] + fmax(elp1[2], elp1[3]);
    y2_min = elp2[1] - fmax(elp2[2], elp2[3]); y2_max = elp2[1] + fmax(elp2[2], elp2[3]);
    y_min = floor(fmax(y1_min, y2_min));
    y_max = ceil(fmin(y1_max, y2_max));

    double search_step = 0.2;
    double S12 = 0;

    for (double i = y_min; i <= y_max+1e-6; i = i + search_step)
    {
        double x11, x12, x21, x22;
        CalculateRangeAtY(parm1, i, &x11, &x12);
        CalculateRangeAtY(parm2, i, &x21, &x22);

        //mexPrintf("[%.4f,%.4f],[%.4f,%.4f]\n", x11, x12, x21, x22);

        if (x11 <= x12&& x21 <= x22)
        {
            if (x11 <= x21 && x12 >= x21)
            {
                if (x12 < x22)
                {
                    S12 += x12 - x21;
                }
                else
                {
                    S12 += x22 - x21;
                }
            }
            else if (x21 <= x11 && x22 >= x11)
            {
                if (x22 < x12)
                {
                    S12 += x22 - x11;
                }
                else
                {
                    S12 += x12 - x11;
                }
            }
        }

    }

    //mexPrintf("%.4f\n", S12);
    *_ration = S12 *search_step / (PI*elp1[2] * elp1[3]/4 + PI*elp2[2] * elp2[3]/4 - S12*search_step);

}